#include "hip/hip_runtime.h"
//This program is written by Abubakr Shafique (abubakr.shafique@gmail.com) 
#include "hip/hip_runtime.h"
#include ""
#include "Equalization_CUDA.h"

__global__ void Calculate_Min_Max(unsigned char* Image, int Channels, int* Min, int* Max);
__global__ void Histogram_Equalization(unsigned char* Image, int Channels, int* Min, int* Max);
__device__ int New_Pixel_Value(int Value, int Min, int Max);

void Histogram_Equalization_CUDA(unsigned char* Image, int Height, int Width, int Channels){
	unsigned char* Dev_Image = NULL;
	int* Dev_Min = NULL;
	int* Dev_Max = NULL;

	//allocate cuda variable memory
	hipMalloc((void**)&Dev_Image, Height * Width * Channels);
	hipMalloc((void**)&Dev_Min, Channels * sizeof(int));
	hipMalloc((void**)&Dev_Max, Channels * sizeof(int));

	int Min[3] = {255, 255, 255};
	int Max[3] = {0, 0, 0};

	//copy CPU data to GPU
	hipMemcpy(Dev_Image, Image, Height * Width * Channels, hipMemcpyHostToDevice);
	hipMemcpy(Dev_Min, Min, Channels * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Dev_Max, Max, Channels * sizeof(int), hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	Calculate_Min_Max << <Grid_Image, 1 >> >(Dev_Image, Channels, Dev_Min, Dev_Max);
	Histogram_Equalization << <Grid_Image, 1 >> >(Dev_Image, Channels, Dev_Min, Dev_Max);

	//copy memory back to CPU from GPU
	hipMemcpy(Image, Dev_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	//free up the memory of GPU
	hipFree(Dev_Image);
}

__global__ void Calculate_Min_Max(unsigned char* Image, int Channels, int* Min, int* Max){
	int x = blockIdx.x;
	int y = blockIdx.y;

	int Image_Idx = (x + y * gridDim.x) * Channels;
	for (int i = 0; i < Channels; i++){
		atomicMin(&Min[i], Image[Image_Idx + i]);
		atomicMax(&Max[i], Image[Image_Idx + i]);
	}
}

__global__ void Histogram_Equalization(unsigned char* Image, int Channels, int* Min, int* Max){
	int x = blockIdx.x;
	int y = blockIdx.y;

	int Image_Idx = (x + y * gridDim.x) * Channels;
	for (int i = 0; i < Channels; i++){
		Image[Image_Idx + i] = New_Pixel_Value(Image[Image_Idx + i], Min[i], Max[i]);
	}
}

__device__ int New_Pixel_Value(int Value, int Min, int Max){
	int Target_Min = 0;
	int Target_Max = 255;

	return (Target_Min + (Value - Min) * (int)((Target_Max - Target_Min)/(Max - Min)));
}
